#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <vector>

#include "../../include/encodes/encode.cuh"
#include "../../include/rgbxy.cuh"
#include "../../include/rgbf.cuh"
#include "../../include/utils.cuh"

namespace CuBlurHash
{
    const char* hash_chars = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz#$%*+,-.:;=?@[]^_{|}~";

    thrust::host_vector<RGBf> get_factors(
        int const& x_components,
        int const& y_components,
        int const& width,
        int const& height,
        thrust::device_vector<RGBXY> const& d_rgb_vector
        );

    std::string encode_factors(
        int const& x_components,
        int const& y_components,
        thrust::host_vector<RGBf> const& factors
        );

    __host__ __device__ inline int encode_dc(RGBf const& rgb);
    __host__ __device__ inline int encode_ac(RGBf const& rgb, float max_value);
    std::string encode_int(
        int value,
        int length
        );

    std::string encode_image(
        CuBlurHash::Image const& image,
        int const& x_components, 
        int const& y_components
        )
    {
        auto h_rgb_vector = image.get_pixels();
        int width = image.get_width();
        int height = image.get_height();

        thrust::device_vector<RGBXY> d_rgb_vector = h_rgb_vector;

        auto factors = get_factors(x_components, y_components, width, height, d_rgb_vector);

        return encode_factors(x_components, y_components, factors);
    }

    struct basis : public std::unary_function<RGBXY, RGBf>
    {
        int x_components;
        int y_components;
        int width;
        int height;

        basis(int const& x_components, int const& y_components, int const& width, int const& height)
            : x_components(x_components), y_components(y_components), width(width), height(height)
        {
        }

        __host__ __device__ RGBf operator()(RGBXY const& rgbxy) const
        {
            RGBf rgbf = sRGB_to_linear(rgbxy.rgb);

            return rgbf * get_basis(rgbxy.x, rgbxy.y);
        }

        __host__ __device__ inline float get_basis(float const& x, float const& y) const
        {
            return cosf(M_PI * x_components * x / width)
                * cosf(M_PI * y_components * y / height);
        }
    };

    RGBf multiply_basis_function(
        int const& x_component,
        int const& y_component,
        int const& width,
        int const& height,
        thrust::device_vector<RGBXY> const& d_rgb_vector
        )
    {
        RGBf result = thrust::transform_reduce(
            d_rgb_vector.begin(),
            d_rgb_vector.end(),
            basis(x_component, y_component, width, height),
            RGBf(),
            thrust::plus<RGBf>()
            );

        float normalisation = (x_component == 0 && y_component == 0) ? 1.0f : 2.0f;
        float scale = normalisation / (width * height);

        return result * scale;
    }

    thrust::host_vector<RGBf> get_factors(
        int const& x_components,
        int const& y_components,
        int const& width,
        int const& height,
        thrust::device_vector<RGBXY> const& d_rgb_vector
        )
    {
        thrust::host_vector<RGBf> h_basis_vector(x_components * y_components);

        for(int x = 0; x < x_components; ++x)
        {
            for(int y = 0; y < y_components; ++y)
            {
                int index = x * y_components + y;
                h_basis_vector[index] = multiply_basis_function(x, y, width, height, d_rgb_vector);
            }
        }

        return h_basis_vector;
    }

    struct max_rgbf_component : public std::unary_function<RGBf, float>
    {
        __host__ __device__ float operator()(RGBf const& rgbf) const
        {
            return fmaxf(fmaxf(fabsf(rgbf.r), fabsf(rgbf.g)), fabsf(rgbf.b));
        }
    };

    std::string encode_factors(
        int const& x_components,
        int const& y_components,
        thrust::host_vector<RGBf> const& factors
        )
    {
        std::string hash = std::string();

        // encode size
        int size_flag = (x_components - 1) + (y_components - 1) * 9;

        hash += encode_int(size_flag, 1);

        // TODO: 1x1 component

        // encode max value
        float max_component = thrust::transform_reduce(
            factors.begin(),
            factors.end(),
            max_rgbf_component(),
            0.0f,
            thrust::maximum<float>()
            );
        int quantised_max_component = fmaxf(0, fminf(82, floorf(max_component * 166.0f - 0.5f)));
        float max_value = (quantised_max_component + 1) / 166.0f;

        hash += encode_int(quantised_max_component, 1);

        // encode factors
        // encode dc
        hash += encode_int(encode_dc(factors[0]), 4);

        // encode ac
        for(int i = 1; i < factors.size(); ++i)
        {
            hash += encode_int(encode_ac(factors[i], max_value), 2);
        }

        return hash;
    }

    __host__ __device__ inline int encode_dc(RGBf const& rgb)
    {
        RGBi rgbi = linear_to_sRGB(rgb);
        return rgbi.to_int();
    }

    __host__ __device__ inline float signed_pow(float base, float exponent)
    {
        return copysignf(powf(fabsf(base), exponent), base);
    }

    __host__ __device__ inline int encode_ac_part(float part, float max_value)
    {
        return fmaxf(0,
                fminf(18.0f,
                    floorf(
                        signed_pow(part / max_value, 0.5f)
                        * 9.0f + 9.5f
                    )
                )
            );
    }

    __host__ __device__ inline int encode_ac(RGBf const& rgb, float max_value)
    {
        int quant_r = encode_ac_part(rgb.r, max_value);
        int quant_g = encode_ac_part(rgb.g, max_value);
        int quant_b = encode_ac_part(rgb.b, max_value);

        return quant_r * 19 * 19 + quant_g * 19 + quant_b;
    }

    std::string encode_int(
        int value,
        int length
        )
    {
        std::string encoded = std::string();
        int divisor = 1;

        for(int i = 0; i < length - 1; ++i)
            divisor *= 83;

        for(int i = 0; i < length; ++i)
        {
            int digit = (value / divisor) % 83;
            divisor /= 83;
            encoded += hash_chars[digit];
        }

        return encoded;
    }
}
